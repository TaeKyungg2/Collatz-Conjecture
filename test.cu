
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
    printf("Hello from thread %d\n", threadIdx.x);
}

int main() {
    kernel<<<1, 5>>>();
    hipDeviceSynchronize();
    return 0;
}
